#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "backward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Backward pass for conversion of spherical harmonics to RGB for
// each Gaussian.
__device__ void computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3 *means, glm::vec3 campos, const float *shs, const bool *clamped, const glm::vec4 *dL_dcolor, glm::vec3 *dL_dmeans, glm::vec4 *dL_dshs)
{
	// Compute intermediate values, as it is done during forward
	glm::vec3 pos = means[idx];
	glm::vec3 dir_orig = pos - campos;
	glm::vec3 dir = dir_orig / glm::length(dir_orig);

	glm::vec4 *sh = ((glm::vec4 *)shs) + idx * max_coeffs;

	// Use PyTorch rule for clamping: if clamping was applied,
	// gradient becomes 0.
	glm::vec4 dL_dRGB = dL_dcolor[idx];
	dL_dRGB.x *= clamped[4 * idx + 0] ? 0 : 1;
	dL_dRGB.y *= clamped[4 * idx + 1] ? 0 : 1;
	dL_dRGB.z *= clamped[4 * idx + 2] ? 0 : 1;
	dL_dRGB.w *= clamped[4 * idx + 3] ? 0 : 1;

	glm::vec4 dRGBdx(0, 0, 0, 0);
	glm::vec4 dRGBdy(0, 0, 0, 0);
	glm::vec4 dRGBdz(0, 0, 0, 0);
	float x = dir.x;
	float y = dir.y;
	float z = dir.z;

	// Target location for this Gaussian to write SH gradients to
	glm::vec4 *dL_dsh = dL_dshs + idx * max_coeffs;

	// No tricks here, just high school-level calculus.
	float dRGBdsh0 = SH_C0;
	dL_dsh[0] = dRGBdsh0 * dL_dRGB;
	if (deg > 0)
	{
		float dRGBdsh1 = -SH_C1 * y;
		float dRGBdsh2 = SH_C1 * z;
		float dRGBdsh3 = -SH_C1 * x;
		dL_dsh[1] = dRGBdsh1 * dL_dRGB;
		dL_dsh[2] = dRGBdsh2 * dL_dRGB;
		dL_dsh[3] = dRGBdsh3 * dL_dRGB;

		dRGBdx = -SH_C1 * sh[3];
		dRGBdy = -SH_C1 * sh[1];
		dRGBdz = SH_C1 * sh[2];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;

			float dRGBdsh4 = SH_C2[0] * xy;
			float dRGBdsh5 = SH_C2[1] * yz;
			float dRGBdsh6 = SH_C2[2] * (2.f * zz - xx - yy);
			float dRGBdsh7 = SH_C2[3] * xz;
			float dRGBdsh8 = SH_C2[4] * (xx - yy);
			dL_dsh[4] = dRGBdsh4 * dL_dRGB;
			dL_dsh[5] = dRGBdsh5 * dL_dRGB;
			dL_dsh[6] = dRGBdsh6 * dL_dRGB;
			dL_dsh[7] = dRGBdsh7 * dL_dRGB;
			dL_dsh[8] = dRGBdsh8 * dL_dRGB;

			dRGBdx += SH_C2[0] * y * sh[4] + SH_C2[2] * 2.f * -x * sh[6] + SH_C2[3] * z * sh[7] + SH_C2[4] * 2.f * x * sh[8];
			dRGBdy += SH_C2[0] * x * sh[4] + SH_C2[1] * z * sh[5] + SH_C2[2] * 2.f * -y * sh[6] + SH_C2[4] * 2.f * -y * sh[8];
			dRGBdz += SH_C2[1] * y * sh[5] + SH_C2[2] * 2.f * 2.f * z * sh[6] + SH_C2[3] * x * sh[7];

			if (deg > 2)
			{
				float dRGBdsh9 = SH_C3[0] * y * (3.f * xx - yy);
				float dRGBdsh10 = SH_C3[1] * xy * z;
				float dRGBdsh11 = SH_C3[2] * y * (4.f * zz - xx - yy);
				float dRGBdsh12 = SH_C3[3] * z * (2.f * zz - 3.f * xx - 3.f * yy);
				float dRGBdsh13 = SH_C3[4] * x * (4.f * zz - xx - yy);
				float dRGBdsh14 = SH_C3[5] * z * (xx - yy);
				float dRGBdsh15 = SH_C3[6] * x * (xx - 3.f * yy);
				dL_dsh[9] = dRGBdsh9 * dL_dRGB;
				dL_dsh[10] = dRGBdsh10 * dL_dRGB;
				dL_dsh[11] = dRGBdsh11 * dL_dRGB;
				dL_dsh[12] = dRGBdsh12 * dL_dRGB;
				dL_dsh[13] = dRGBdsh13 * dL_dRGB;
				dL_dsh[14] = dRGBdsh14 * dL_dRGB;
				dL_dsh[15] = dRGBdsh15 * dL_dRGB;

				dRGBdx += (SH_C3[0] * sh[9] * 3.f * 2.f * xy +
						   SH_C3[1] * sh[10] * yz +
						   SH_C3[2] * sh[11] * -2.f * xy +
						   SH_C3[3] * sh[12] * -3.f * 2.f * xz +
						   SH_C3[4] * sh[13] * (-3.f * xx + 4.f * zz - yy) +
						   SH_C3[5] * sh[14] * 2.f * xz +
						   SH_C3[6] * sh[15] * 3.f * (xx - yy));

				dRGBdy += (SH_C3[0] * sh[9] * 3.f * (xx - yy) +
						   SH_C3[1] * sh[10] * xz +
						   SH_C3[2] * sh[11] * (-3.f * yy + 4.f * zz - xx) +
						   SH_C3[3] * sh[12] * -3.f * 2.f * yz +
						   SH_C3[4] * sh[13] * -2.f * xy +
						   SH_C3[5] * sh[14] * -2.f * yz +
						   SH_C3[6] * sh[15] * -3.f * 2.f * xy);

				dRGBdz += (SH_C3[1] * sh[10] * xy +
						   SH_C3[2] * sh[11] * 4.f * 2.f * yz +
						   SH_C3[3] * sh[12] * 3.f * (2.f * zz - xx - yy) +
						   SH_C3[4] * sh[13] * 4.f * 2.f * xz +
						   SH_C3[5] * sh[14] * (xx - yy));
			}
		}
	}

	// The view direction is an input to the computation. View direction
	// is influenced by the Gaussian's mean, so SHs gradients
	// must propagate back into 3D position.
	glm::vec3 dL_ddir(glm::dot(dRGBdx, dL_dRGB), glm::dot(dRGBdy, dL_dRGB), glm::dot(dRGBdz, dL_dRGB));

	// Account for normalization of direction
	float3 dL_dmean = dnormvdv(float3{dir_orig.x, dir_orig.y, dir_orig.z}, float3{dL_ddir.x, dL_ddir.y, dL_ddir.z});

	// Gradients of loss w.r.t. Gaussian means, but only the portion
	// that is caused because the mean affects the view-dependent color.
	// Additional mean gradient is accumulated in below methods.
	dL_dmeans[idx] += glm::vec3(dL_dmean.x, dL_dmean.y, dL_dmean.z);
}

// Backward version of the rendering procedure.
template <uint32_t C>
__global__ void __launch_bounds__(BLOCK_X *BLOCK_Y)
	renderCUDA(
		const uint2 *__restrict__ ranges,
		const uint32_t *__restrict__ point_list,
		const int S, int W, int H,
		const float *__restrict__ bg_color,
		const float2 *__restrict__ points_xy_image,
		const float4 *__restrict__ normal_opacity,
		const float *__restrict__ transMats,
		const float *__restrict__ colors,
		const float *__restrict__ depths,
		const float *__restrict__ features,
		const float *__restrict__ final_Ts,
		const int32_t *__restrict__ n_contrib,
		const float *__restrict__ dL_dpixels,
		const float *__restrict__ dL_depths,
		const float *__restrict__ dL_masks,
		const float *__restrict__ dL_dpix_feature,
		float *__restrict__ dL_dtransMat,
		float4 *__restrict__ dL_dmean2D,
		// float4 *__restrict__ dL_dconic2D,
		float *__restrict__ dL_dopacity,
		float *__restrict__ dL_dcolors,
		float *__restrict__ dL_dfeatures,
		float *__restrict__ dL_dnormals,
		const float vfov_min,
		const float vfov_max,
		const float hfov_min,
		const float hfov_max,
		const float scale_factor)
{
	// We rasterize again. Compute necessary block info.
	auto block = cg::this_thread_block();
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	const uint2 pix_min = {block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y};
	const uint2 pix_max = {min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y, H)};
	const uint2 pix = {pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y};
	const uint32_t pix_id = W * pix.y + pix.x;
	const float2 pixf = {(float)pix.x, (float)pix.y};

	const bool inside = pix.x < W && pix.y < H;
	const uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];

	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);

	bool done = !inside;
	int toDo = range.y - range.x;

	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_normal_opacity[BLOCK_SIZE];
	__shared__ float collected_colors[C * BLOCK_SIZE];
	__shared__ float collected_depths[BLOCK_SIZE];
	__shared__ float collected_features[13 * BLOCK_SIZE];
	__shared__ float3 collected_Tu[BLOCK_SIZE];
	__shared__ float3 collected_Tv[BLOCK_SIZE];
	__shared__ float3 collected_Tw[BLOCK_SIZE];

	// In the forward, we stored the final value for T, the
	// product of all (1 - alpha) factors.
	const float T_final = inside ? final_Ts[pix_id] : 0;
	float T = T_final;

	// We start from the back. The ID of the last contributing
	// Gaussian is known from each pixel from the forward.
	uint32_t contributor = toDo;
	const int last_contributor = inside ? n_contrib[pix_id] : 0;
	const int median_contributor = inside ? n_contrib[pix_id + H * W] : 0;

	float accum_rec[C] = {0};
	float accum_feature_rec[13 + 3] = {0};
	float dL_dpixel[C] = {0};
	float dL_dfeature[13] = {0};
	float dL_depth = 0;
	float dL_dmedian_depth = 0;
	float dL_ddistortion = 0;
	float dL_depth_square = 0;
	float dL_mask = 0;
	float accum_depth_rec = 0;
	float accum_mask_rec = 0;

	// for compute gradient with respect to the distortion map
	const float final_D = inside ? final_Ts[pix_id + H * W] : 0;
	const float final_D2 = inside ? final_Ts[pix_id + 2 * H * W] : 0;
	const float final_A = 1 - T_final;
	float last_dL_dT = 0;

	if (inside)
	{
		for (int i = 0; i < C; i++)
		{
			dL_dpixel[i] = dL_dpixels[i * H * W + pix_id];
		}
		for (int i = 0; i < S + 3; i++)
		{
			dL_dfeature[i] = dL_dpix_feature[i * H * W + pix_id];
		}
		dL_depth = dL_depths[pix_id];
		dL_dmedian_depth = dL_depths[H * W + pix_id];
		dL_ddistortion = dL_depths[2 * H * W + pix_id];
		dL_depth_square = dL_depths[3 * H * W + pix_id];
		dL_mask = dL_masks[pix_id];
	}

	float last_alpha = 0;
	float last_color[C] = {0};
	float last_feature[13] = {0};
	float last_depth = 0;

	// Gradient of pixel coordinate w.r.t. normalized
	// screen-space viewport corrdinates (-1 to 1)
	// const float ddelx_dx = 0.5 * W;
	// const float ddely_dy = 0.5 * H;

	// vfov 角度制转弧度制
	const float VFOV_max = MY_PI / 2 - vfov_min * MY_PI / 180;
	const float VFOV_min = MY_PI / 2 - vfov_max * MY_PI / 180;

	// hfov 角度制转弧度制
	const float HFOV_max = hfov_max * MY_PI / 180;
	const float HFOV_min = hfov_min * MY_PI / 180;

	const float near = near_n * scale_factor;
	const float far = far_n * scale_factor;

	// Traverse all Gaussians
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// Load auxiliary data into shared memory, start in the BACK
		// and load them in revers order.
		block.sync();
		const int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			const int coll_id = point_list[range.y - progress - 1];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_depths[block.thread_rank()] = depths[coll_id];
			collected_normal_opacity[block.thread_rank()] = normal_opacity[coll_id];
			collected_Tu[block.thread_rank()] = {transMats[9 * coll_id + 0], transMats[9 * coll_id + 1], transMats[9 * coll_id + 2]};
			collected_Tv[block.thread_rank()] = {transMats[9 * coll_id + 3], transMats[9 * coll_id + 4], transMats[9 * coll_id + 5]};
			collected_Tw[block.thread_rank()] = {transMats[9 * coll_id + 6], transMats[9 * coll_id + 7], transMats[9 * coll_id + 8]};
			for (int i = 0; i < C; i++)
				collected_colors[i * BLOCK_SIZE + block.thread_rank()] = colors[coll_id * C + i];
			for (int i = 0; i < S; i++)
				collected_features[i * BLOCK_SIZE + block.thread_rank()] = features[coll_id * S + i];
		}
		block.sync();

		// Iterate over Gaussians
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current Gaussian ID. Skip, if this one
			// is behind the last contributor for this pixel.
			contributor--;
			if (contributor >= last_contributor)
				continue;

			// Compute blending values, as before.
			const float2 xy = collected_xy[j];
			const float3 Tu = collected_Tu[j];
			const float3 Tv = collected_Tv[j];
			const float3 Tw = collected_Tw[j];

			const float phi = pixf.x * (HFOV_max - HFOV_min) / W + HFOV_min;
			const float theta = pixf.y * (VFOV_max - VFOV_min) / H + VFOV_min;

			float3 k = cos(phi) * Tu - sin(phi) * Tw;
			float3 l = sin(phi) * cos(theta) * Tu + sin(theta) * Tv + cos(phi) * cos(theta) * Tw;
			float3 p = cross(k, l);
			if (p.z == 0.0)
				continue;
			float2 s = {p.x / p.z, p.y / p.z};
			float rho3d = (s.x * s.x + s.y * s.y);
			float2 d = {xy.x - pixf.x, xy.y - pixf.y};
			float rho2d = FilterInvSquare * (d.x * d.x + d.y * d.y);

			// if (rho3d > rho2d)
			// 	continue;

			float rho = min(rho3d, rho2d);
			// cos(theta) 会趋于0，导致不稳定
			// const float depth_3d_0 = -(s.x * Tv.x + s.y * Tv.y + Tv.z) / cos(theta);
			const float s_Tu = s.x * Tu.x + s.y * Tu.y + Tu.z;
			const float s_Tv = s.x * Tv.x + s.y * Tv.y + Tv.z;
			const float s_Tw = s.x * Tw.x + s.y * Tw.y + Tw.z;
			const float depth_3d = s_Tu * sin(theta) * sin(phi) - s_Tv * cos(theta) + s_Tw * sin(theta) * cos(phi);
			const float depth = (rho3d <= rho2d) ? depth_3d : collected_depths[j];
			if (depth < near || depth > far)
				continue;
			float4 nor_o = collected_normal_opacity[j];
			float normal[3] = {nor_o.x, nor_o.y, nor_o.z};
			float opa = nor_o.w;

			const float power = -0.5f * rho;
			if (power > 0.0f)
				continue;

			const float G = exp(power);
			const float alpha = min(0.99f, opa * G);
			if (alpha < 1.0f / 255.0f)
				continue;

			T = T / (1.f - alpha);
			const float dchannel_dcolor = alpha * T;

			// Propagate gradients to per-Gaussian colors and keep
			// gradients w.r.t. alpha (blending factor for a Gaussian/pixel
			// pair).
			float dL_dalpha = 0.0f;
			const int global_id = collected_id[j];
			for (int ch = 0; ch < C; ch++)
			{
				const float c = collected_colors[ch * BLOCK_SIZE + j];
				// Update last color (to be used in the next iteration)
				accum_rec[ch] = last_alpha * last_color[ch] + (1.f - last_alpha) * accum_rec[ch];
				last_color[ch] = c;

				const float dL_dchannel = dL_dpixel[ch];
				dL_dalpha += (c - accum_rec[ch]) * dL_dchannel;
				// Update the gradients w.r.t. color of the Gaussian.
				// Atomic, since this pixel is just one of potentially
				// many that were affected by this Gaussian.
				atomicAdd(&(dL_dcolors[global_id * C + ch]), dchannel_dcolor * dL_dchannel);
			}

			float dL_dr = 0.0f;
			dL_dr += alpha * T * dL_depth;
			dL_dr += alpha * T * 2 * depth * dL_depth_square;
			if (contributor == median_contributor - 1)
			{
				dL_dr += dL_dmedian_depth;
			}

			float dL_dweight = 0;
			const float m_d = far / (far - near) * (1 - near / depth);
			const float dmd_dd = (far * near) / ((far - near) * depth * depth);
			dL_dweight += (final_D2 + m_d * m_d * final_A - 2 * m_d * final_D) * dL_ddistortion;
			dL_dalpha += dL_dweight - last_dL_dT;
			// propagate the current weight W_{i} to next weight W_{i-1}
			last_dL_dT = dL_dweight * alpha + (1 - alpha) * last_dL_dT;
			const float dL_dmd = 2.0f * (T * alpha) * (m_d * final_A - final_D) * dL_ddistortion;
			dL_dr += dL_dmd * dmd_dd;

			for (int ch = 0; ch < S + 3; ch++)
			{
				float feature;
				if (ch < S)
					feature = collected_features[ch * BLOCK_SIZE + j];
				else
					feature = normal[ch - S];

				// Update last color (to be used in the next iteration)
				accum_feature_rec[ch] = last_alpha * last_feature[ch] + (1.f - last_alpha) * accum_feature_rec[ch];
				last_feature[ch] = feature;

				const float dL_dchannelfeature = dL_dfeature[ch];
				// dL_dalpha += (feature - accum_feature_rec[ch]) * dL_dchannelfeature;
				// Update the gradients w.r.t. color of the Gaussian.
				// Atomic, since this pixel is just one of potentially
				// many that were affected by this Gaussian.
				if (ch < S)
					atomicAdd(&(dL_dfeatures[global_id * S + ch]), dchannel_dcolor * dL_dchannelfeature);
				else
				{
					dL_dalpha += (feature - accum_feature_rec[ch]) * dL_dchannelfeature;
					atomicAdd(&(dL_dnormals[global_id * 3 + ch - S]), dchannel_dcolor * dL_dchannelfeature); // normal loss
				}
			}

			// Propagate gradients to per-Gaussian depths
			accum_depth_rec = last_alpha * last_depth + (1.f - last_alpha) * accum_depth_rec;
			last_depth = depth;
			dL_dalpha += ((depth - accum_depth_rec) * dL_depth);

			// Propagate gradients from masks
			accum_mask_rec = last_alpha + (1.f - last_alpha) * accum_mask_rec;
			dL_dalpha += ((1.0 - accum_mask_rec) * dL_mask);

			dL_dalpha *= T;
			// Update last alpha (to be used in the next iteration)
			last_alpha = alpha;

			// Account for fact that alpha also influences how much of
			// the background color is added if nothing left to blend
			float bg_dot_dpixel = 0;
			for (int i = 0; i < C; i++)
				bg_dot_dpixel += bg_color[i] * dL_dpixel[i];
			dL_dalpha += (-T_final / (1.f - alpha)) * bg_dot_dpixel;

			// Helpful reusable temporary variables
			const float dL_dG = nor_o.w * dL_dalpha;

			if (rho3d <= rho2d)
			{
				// Update gradients w.r.t. covariance of Gaussian 3x3 (T)
				const float2 dL_ds = {
					dL_dG * -G * s.x + dL_dr * (Tu.x * sin(theta) * sin(phi) - Tv.x * cos(theta) + Tw.x * sin(theta) * cos(phi)),
					dL_dG * -G * s.y + dL_dr * (Tu.y * sin(theta) * sin(phi) - Tv.y * cos(theta) + Tw.y * sin(theta) * cos(phi))};
				const float3 dr_dTu = sin(theta) * sin(phi) * float3{s.x, s.y, 1.f};
				const float3 dr_dTv = -cos(theta) * float3{s.x, s.y, 1.0};
				const float3 dr_dTw = sin(theta) * cos(phi) * float3{s.x, s.y, 1.f};
				const float dsx_pz = dL_ds.x / p.z;
				const float dsy_pz = dL_ds.y / p.z;
				const float3 dL_dp = {dsx_pz, dsy_pz, -(dsx_pz * s.x + dsy_pz * s.y)};
				const float3 dL_dk = cross(l, dL_dp);
				const float3 dL_dl = cross(dL_dp, k);

				const float3 dL_dTu = cos(phi) * dL_dk + sin(phi) * cos(theta) * dL_dl + dL_dr * dr_dTu;
				const float3 dL_dTv = sin(theta) * dL_dl + dL_dr * dr_dTv;
				const float3 dL_dTw = -sin(phi) * dL_dk + cos(phi) * cos(theta) * dL_dl + dL_dr * dr_dTw;

				// Update gradients w.r.t. 3D covariance (3x3 matrix)
				atomicAdd(&dL_dtransMat[global_id * 9 + 0], dL_dTu.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 1], dL_dTu.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 2], dL_dTu.z);
				atomicAdd(&dL_dtransMat[global_id * 9 + 3], dL_dTv.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 4], dL_dTv.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 5], dL_dTv.z);
				atomicAdd(&dL_dtransMat[global_id * 9 + 6], dL_dTw.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 7], dL_dTw.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 8], dL_dTw.z);

				// const float u = transMats[global_id * 9 + 2];
				// const float v = transMats[global_id * 9 + 5];
				// const float w = transMats[global_id * 9 + 8];
				// const float dL_du = dL_dTu.z;
				// const float dL_dv = dL_dTv.z;
				// const float dL_dw = dL_dTw.z;

				// const float dx_du = w / (u * u + w * w);
				// const float dx_dw = -u / (u * u + w * w);
				// const float dL_dmean2D_x = (dL_du / dx_du + dL_dw / dx_dw) * 0.5 * (HFOV_max - HFOV_min);

				// const float dy_du = -u * v / (sqrt(u * u + w * w) * (u * u + v * v + w * w));
				// const float dy_dv = sqrt(u * u + w * w) / (u * u + v * v + w * w);
				// const float dy_dw = -w * v / (sqrt(u * u + w * w) * (u * u + v * v + w * w));
				// const float dL_dmean2D_y = (dL_du / dy_du + dL_dv / dy_dv + dL_dw / dy_dw) * 0.5 * (VFOV_max - VFOV_min);

				// const float phi = atan2f(u, w);
				// // const float theta = atan2f(sqrt(u * u + w * w), -v);
				// // const float r = sqrt(u * u + v * v + w * w);

				// const float du_dphi = w;  // r * sin(theta) * cos(phi)
				// const float dw_dphi = -u; // -r * sin(theta) * sin(phi)
				// const float dL_dmean2D_x = (dL_du * du_dphi + dL_dw * dw_dphi) * 0.5 * (HFOV_max - HFOV_min);

				// const float du_dtheta = -v * sin(phi);		 // r * cos(theta) * sin(phi)
				// const float dv_dtheta = sqrt(u * u + w * w); // r * sin(theta)
				// const float dw_dtheta = -v * cos(phi);		 // r * cos(theta) * cos(phi)
				// const float dL_dmean2D_y = (dL_du * du_dtheta + dL_dv * dv_dtheta + dL_dw * dw_dtheta) * 0.5 * (VFOV_max - VFOV_min);

				// // AbsGS densitify
				// atomicAdd(&dL_dmean2D[global_id].z, fabs(dL_dmean2D_x));
				// atomicAdd(&dL_dmean2D[global_id].w, fabs(dL_dmean2D_y));
			}
			else
			{
				const float dG_ddelx = -G * FilterInvSquare * d.x;
				const float dG_ddely = -G * FilterInvSquare * d.y;

				// Update gradients w.r.t. 2D mean position of the Gaussian
				atomicAdd(&dL_dmean2D[global_id].x, dL_dG * dG_ddelx);
				atomicAdd(&dL_dmean2D[global_id].y, dL_dG * dG_ddely);
				atomicAdd(&dL_dtransMat[global_id * 9 + 2], dL_dr * Tu.z / depth);
				atomicAdd(&dL_dtransMat[global_id * 9 + 5], dL_dr * Tv.z / depth);
				atomicAdd(&dL_dtransMat[global_id * 9 + 8], dL_dr * Tw.z / depth);

				// // AbsGS densitify
				// atomicAdd(&dL_dmean2D[global_id].z, fabs(dL_dG * dG_ddelx * 0.5 * W));
				// atomicAdd(&dL_dmean2D[global_id].w, fabs(dL_dG * dG_ddely * 0.5 * H));
			}

			// Update gradients w.r.t. opacity of the Gaussian
			atomicAdd(&(dL_dopacity[global_id]), G * dL_dalpha);
		}
	}
}

__device__ void compute_transmat_aabb(
	int idx,
	const float3 *p_origs,
	const glm::vec3 *scales,
	const glm::vec4 *rots,
	const float *projmatrix,
	const float *viewmatrix,
	const int W, const int H,
	const float3 *dL_dnormals,
	const float4 *dL_dmean2Ds,
	float *dL_dTs,
	glm::vec3 *dL_dmeans,
	glm::vec3 *dL_dscales,
	glm::vec4 *dL_drots,
	const float VFOV_max,
	const float VFOV_min,
	const float HFOV_max,
	const float HFOV_min)
{
	glm::mat3 T;
	float3 normal;
	glm::mat3x4 P;
	glm::mat3 R;
	glm::mat3 S;
	float3 p_orig;
	glm::vec4 rot;
	glm::vec3 scale;

	// Get transformation matrix of the Gaussian
	p_orig = p_origs[idx];
	rot = rots[idx];
	scale = scales[idx];
	R = quat_to_rotmat(rot);
	S = scale_to_mat(scale, 1.0f);

	glm::mat3 L = R * S;
	glm::mat3x4 M = glm::mat3x4(
		glm::vec4(L[0], 0.0),
		glm::vec4(L[1], 0.0),
		glm::vec4(p_orig.x, p_orig.y, p_orig.z, 1));

	glm::mat4 world2camera = glm::mat4(
		viewmatrix[0], viewmatrix[4], viewmatrix[8], viewmatrix[12],
		viewmatrix[1], viewmatrix[5], viewmatrix[9], viewmatrix[13],
		viewmatrix[2], viewmatrix[6], viewmatrix[10], viewmatrix[14],
		viewmatrix[3], viewmatrix[7], viewmatrix[11], viewmatrix[15]);

	glm::mat3x4 mat4x3_to_mat3 = glm::mat3x4(
		1, 0, 0, 0,
		0, 1, 0, 0,
		0, 0, 1, 0);

	P = world2camera * mat4x3_to_mat3;
	T = glm::transpose(M) * P;
	normal = transformVec4x3({L[2].x, L[2].y, L[2].z}, viewmatrix);

	// Update gradients w.r.t. transformation matrix of the Gaussian
	glm::mat3 dL_dT = glm::mat3(
		dL_dTs[idx * 9 + 0], dL_dTs[idx * 9 + 1], dL_dTs[idx * 9 + 2],
		dL_dTs[idx * 9 + 3], dL_dTs[idx * 9 + 4], dL_dTs[idx * 9 + 5],
		dL_dTs[idx * 9 + 6], dL_dTs[idx * 9 + 7], dL_dTs[idx * 9 + 8]);
	float4 dL_dmean2D = dL_dmean2Ds[idx];
	if (dL_dmean2D.x != 0 || dL_dmean2D.y != 0)
	{
		const float Wrange = W / (HFOV_max - HFOV_min);
		const float Hrange = H / (VFOV_max - VFOV_min);

		const float u = T[0].z;
		const float v = T[1].z;
		const float w = T[2].z;
		const float r2_uw = u * u + w * w;
		const float r_uw = sqrt(u * u + w * w);
		const float r2 = u * u + v * v + w * w;
		const float r = sqrt(u * u + v * v + w * w);

		dL_dT[0].z += dL_dmean2D.x * Wrange * w / r2_uw - dL_dmean2D.y * Hrange * u * v / (r_uw * r2);
		dL_dT[1].z += dL_dmean2D.y * Hrange * r_uw / r2;
		dL_dT[2].z += -dL_dmean2D.x * Wrange * u / r2_uw - dL_dmean2D.y * Hrange * v * w / (r_uw * r2);
	}

	// Update gradients w.r.t. scaling, rotation, position of the Gaussian
	glm::mat3x4 dL_dM = P * glm::transpose(dL_dT);
	float3 dL_dtn = transformVec4x3Transpose(dL_dnormals[idx], viewmatrix);
#if DUAL_VISIABLE
	float multiplier = normal.z < 0 ? 1 : -1;
	dL_dtn = multiplier * dL_dtn;
#endif
	glm::mat3 dL_dRS = glm::mat3(
		glm::vec3(dL_dM[0]),
		glm::vec3(dL_dM[1]),
		glm::vec3(dL_dtn.x, dL_dtn.y, dL_dtn.z));

	glm::mat3 dL_dR = glm::mat3(
		dL_dRS[0] * glm::vec3(scale.x),
		dL_dRS[1] * glm::vec3(scale.y),
		dL_dRS[2]);

	dL_drots[idx] = quat_to_rotmat_vjp(rot, dL_dR);
	dL_dscales[idx] = glm::vec3(
		(float)glm::dot(dL_dRS[0], R[0]),
		(float)glm::dot(dL_dRS[1], R[1]),
		0);
	dL_dmeans[idx] = glm::vec3(dL_dM[2]);
}

template <int C>
__global__ void preprocessCUDA(
	int P, int D, int M,
	const float3 *means3D,
	const float *transMats,
	const int *radii,
	const float *shs,
	const bool *clamped,
	const glm::vec3 *scales,
	const glm::vec4 *rotations,
	const float scale_modifier,
	const float *viewmatrix,
	const float *projmatrix,
	const float focal_x,
	const float focal_y,
	const float tan_fovx,
	const float tan_fovy,
	const glm::vec3 *campos,
	// grad input
	float *dL_dtransMats,
	const float *dL_dnormal3Ds,
	float *dL_dcolors,
	float *dL_dshs,
	float4 *dL_dmean2Ds,
	glm::vec3 *dL_dmean3Ds,
	glm::vec3 *dL_dscales,
	glm::vec4 *dL_drots,
	const float VFOV_max,
	const float VFOV_min,
	const float HFOV_max,
	const float HFOV_min)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	const int W = int(focal_x * tan_fovx * 2);
	const int H = int(focal_y * tan_fovy * 2);

	compute_transmat_aabb(
		idx,
		means3D, scales, rotations,
		projmatrix, viewmatrix, W, H,
		(float3 *)dL_dnormal3Ds,
		dL_dmean2Ds,
		(dL_dtransMats),
		dL_dmean3Ds,
		dL_dscales,
		dL_drots,
		VFOV_max,
		VFOV_min,
		HFOV_max,
		HFOV_min);

	if (shs)
		computeColorFromSH(idx, D, M, (glm::vec3 *)means3D, *campos, shs, clamped, (glm::vec4 *)dL_dcolors, (glm::vec3 *)dL_dmean3Ds, (glm::vec4 *)dL_dshs);

	// hack the gradient here for densitification
	// float depth = transMats[idx * 9 + 8];
	// dL_dmean2Ds[idx].x = dL_dtransMats[idx * 9 + 2] * depth * 0.5 * float(W); // to ndc
	// dL_dmean2Ds[idx].y = dL_dtransMats[idx * 9 + 5] * depth * 0.5 * float(H); // to ndc

	const float u = transMats[idx * 9 + 2];
	const float v = transMats[idx * 9 + 5];
	const float w = transMats[idx * 9 + 8];
	const float dL_du = dL_dtransMats[idx * 9 + 2];
	const float dL_dv = dL_dtransMats[idx * 9 + 5];
	const float dL_dw = dL_dtransMats[idx * 9 + 8];

	// const float dx_du = w / (u * u + w * w);
	// const float dx_dw = -u / (u * u + w * w);
	// dL_dmean2Ds[idx].x = (dL_du / dx_du + dL_dw / dx_dw) * 0.5 * (HFOV_max - HFOV_min);

	// const float dy_du = -u * v / (sqrt(u * u + w * w) * (u * u + v * v + w * w));
	// const float dy_dv = sqrt(u * u + w * w) / (u * u + v * v + w * w);
	// const float dy_dw = -w * v / (sqrt(u * u + w * w) * (u * u + v * v + w * w));
	// dL_dmean2Ds[idx].y = (dL_du / dy_du + dL_dv / dy_dv + dL_dw / dy_dw) * 0.5 * (VFOV_max - VFOV_min) * W / H;

	const float phi = atan2f(u, w);
	// const float theta = atan2f(sqrt(u * u + w * w), -v);
	// const float r = sqrt(u * u + v * v + w * w);

	const float du_dphi = w;  // r * sin(theta) * cos(phi)
	const float dw_dphi = -u; // -r * sin(theta) * sin(phi)
	dL_dmean2Ds[idx].x = (dL_du * du_dphi + dL_dw * dw_dphi) * 0.5 * (HFOV_max - HFOV_min);

	const float du_dtheta = -v * sin(phi);		 // r * cos(theta) * sin(phi)
	const float dv_dtheta = sqrt(u * u + w * w); // r * sin(theta)
	const float dw_dtheta = -v * cos(phi);		 // r * cos(theta) * cos(phi)
	dL_dmean2Ds[idx].y = (dL_du * du_dtheta + dL_dv * dv_dtheta + dL_dw * dw_dtheta) * 0.5 * (VFOV_max - VFOV_min) * W / H;
}

void BACKWARD::preprocess(
	int P, int D, int M,
	const float3 *means3D,
	const int *radii,
	const float *shs,
	const bool *clamped,
	const glm::vec3 *scales,
	const glm::vec4 *rotations,
	const float scale_modifier,
	const float *transMats,
	const float *viewmatrix,
	const float *projmatrix,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	const glm::vec3 *campos,
	float4 *dL_dmean2D,
	float *dL_dnormals,
	float *dL_dtransMats,
	glm::vec3 *dL_dmean3D,
	float *dL_dcolor,
	float *dL_dsh,
	glm::vec3 *dL_dscale,
	glm::vec4 *dL_drot,
	const float vfov_min,
	const float vfov_max,
	const float hfov_min,
	const float hfov_max,
	const int width, int height)
{
	// vfov 角度制转弧度制
	const float VFOV_max = MY_PI / 2 - vfov_min * MY_PI / 180;
	const float VFOV_min = MY_PI / 2 - vfov_max * MY_PI / 180;

	// hfov 角度制转弧度制
	const float HFOV_max = hfov_max * MY_PI / 180;
	const float HFOV_min = hfov_min * MY_PI / 180;

	preprocessCUDA<NUM_CHANNELS><<<(P + 255) / 256, 256>>>(
		P, D, M,
		(float3 *)means3D,
		transMats,
		radii,
		shs,
		clamped,
		(glm::vec3 *)scales,
		(glm::vec4 *)rotations,
		scale_modifier,
		viewmatrix,
		projmatrix,
		focal_x,
		focal_y,
		tan_fovx,
		tan_fovy,
		campos,
		dL_dtransMats,
		dL_dnormals,
		dL_dcolor,
		dL_dsh,
		dL_dmean2D,
		dL_dmean3D,
		dL_dscale,
		dL_drot,
		VFOV_max,
		VFOV_min,
		HFOV_max,
		HFOV_min);
}

void BACKWARD::render(
	const dim3 grid, const dim3 block,
	const uint2 *ranges,
	const uint32_t *point_list,
	const int S, int W, int H,
	const float *bg_color,
	const float2 *means2D,
	const float4 *normal_opacity,
	const float *colors,
	const float *transMats,
	const float *depths,
	const float *features,
	const float *final_Ts,
	const int32_t *n_contrib,
	const float *dL_dpixels,
	const float *dL_depths,
	const float *dL_masks,
	const float *dL_dpix_feature,
	float *dL_dtransMat,
	float4 *dL_dmean2D,
	float *dL_dopacity,
	float *dL_dcolors,
	float *dL_dfeatures,
	float *dL_dnormals,
	const float vfov_min,
	const float vfov_max,
	const float hfov_min,
	const float hfov_max,
	const float scale_factor)
{
	renderCUDA<NUM_CHANNELS><<<grid, block>>>(
		ranges,
		point_list,
		S, W, H,
		bg_color,
		means2D,
		normal_opacity,
		transMats,
		colors,
		depths,
		features,
		final_Ts,
		n_contrib,
		dL_dpixels,
		dL_depths,
		dL_masks,
		dL_dpix_feature,
		dL_dtransMat,
		dL_dmean2D,
		dL_dopacity,
		dL_dcolors,
		dL_dfeatures,
		dL_dnormals,
		vfov_min,
		vfov_max,
		hfov_min,
		hfov_max,
		scale_factor);
}